#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void update(float *xCurrPos, float *yCurrPos, float *xLastPos, float *yLastPos, float *xAcc, float *yAcc, float xGravity, float yGravity, const float dt, int numElements, const float *radii, bool *collided) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        xAcc[i] += xGravity;
        yAcc[i] += yGravity;


        float VELOCITY_DAMPING = 40.0f;

        const float curr_pos_x = xCurrPos[i];
        const float curr_pos_y = yCurrPos[i];
        float last_update_move_x = curr_pos_x - xLastPos[i];

        float last_update_move_y = curr_pos_y - yLastPos[i];

         float new_position_x =
                curr_pos_x + last_update_move_x + (xAcc[i] - last_update_move_x * VELOCITY_DAMPING) * (dt * dt);
         float new_position_y =
                curr_pos_y + last_update_move_y + (yAcc[i] - last_update_move_y * VELOCITY_DAMPING) * (dt * dt);
        float movex = new_position_x - curr_pos_x;
        float movey = new_position_y - curr_pos_y;
        float max_speed = radii[i];

        if (collided[i]){
            max_speed = radii[i] / 10.f;
        }
        if (movex >max_speed) {
            new_position_x = curr_pos_x +max_speed ;
        }
        if(movex < -max_speed) {
            new_position_x = curr_pos_x - max_speed;
        }
        if(movey > max_speed) {
            new_position_y = curr_pos_y + max_speed;
        }
        if(movey < -max_speed) {
            new_position_y = curr_pos_y - max_speed;
        }

        xLastPos[i] = curr_pos_x;
        yLastPos[i] = curr_pos_y;
        xCurrPos[i] = new_position_x;
        yCurrPos[i] = new_position_y;

        xAcc[i] = 0.0f;
        yAcc[i] = 0.0f;

        float pos_x = xCurrPos[i];
        float pos_y = yCurrPos[i];
        float radius = radii[i];
        if (pos_x - radius < -1.0) {
            xCurrPos[i] = -1.0 + radius;
        } else if (pos_x + radius > 1.0) {
            xCurrPos[i] = 1.0 - radius;
        }

        // Vertical walls
        if (pos_y - radius < -1.0) {
            yCurrPos[i] = -1.0 + radius;
        } else if (pos_y + radius > 1.0) {
            yCurrPos[i] = 1.0 - radius;
        }
    }


}

__global__ void solveContact(float *xPos, float *yPos, float *xPosRes, float *yPosRes, float *radii, int numElements, bool *collided) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;


if (i < numElements && j < numElements && i != j) {
        constexpr float response_coef = 1.10f;
        constexpr float eps           = 0.00001f;
        float pos1_x = xPos[i];
        float pos1_y = yPos[i];
        float pos2_x = xPos[j];
        float pos2_y = yPos[j];
        float radius1 = radii[i];
        float radius2 = radii[j];


        const float dist2 = (pos2_x - pos1_x) * (pos2_x - pos1_x) + (pos2_y - pos1_y) * (pos2_y - pos1_y);
        if (dist2 > eps && dist2 < (radius1 + radius2) * (radius1 + radius2)) {
            const float dist          = sqrt(dist2);
            const float delta  = response_coef * 0.5f * (radius1 + radius2 - dist);
            float col_vec_x = (pos2_x - pos1_x) / dist * delta;
            float col_vec_y = (pos2_y - pos1_y) / dist * delta;

            atomicAdd(xPosRes + i, -col_vec_x);
            atomicAdd(yPosRes + i, -col_vec_y);
            collided[i] = true;
        }



    }
}

extern "C" void cuda_solve_collisions(float *currPositionsX, float *currPositionsY, float *radii,
                                      float *lastPositionsX, float *lastPositionsY, float *accelerationX,
                                      float *accelerationY,  float xGravity,  float yGravity,
                                      float dt, int substeps, int numElements){
    float *d_curr_positions_x = NULL; // input
    float *d_curr_positions_y= NULL; // input
    float *d_radii = NULL; // input
    float *d_last_positions_x = NULL; // input
    float *d_last_positions_y = NULL; // input
    float *d_acceleration_x = NULL; // input
    float *d_acceleration_y = NULL; // input
    float *d_result_x = NULL; // output
    float *d_result_y = NULL; // output
    bool *collided = NULL;


    hipMalloc((void **)&d_curr_positions_x, numElements * sizeof(float));
    hipMalloc((void **)&d_curr_positions_y, numElements * sizeof(float));
    hipMalloc((void **)&d_radii, numElements * sizeof(float));
    hipMalloc((void **)&d_result_x, numElements * sizeof(float));
    hipMalloc((void **)&d_result_y, numElements * sizeof(float));
    hipMalloc((void **)&d_last_positions_x, numElements * sizeof(float));
    hipMalloc((void **)&d_last_positions_y, numElements * sizeof(float));
    hipMalloc((void **)&d_acceleration_x, numElements * sizeof(float));
    hipMalloc((void **)&d_acceleration_y, numElements * sizeof(float));
    hipMalloc((void **)&collided, numElements * sizeof(bool));

    hipMemcpy(d_curr_positions_x, currPositionsX, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_curr_positions_y, currPositionsY, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_radii, radii, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_last_positions_x, lastPositionsX, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_last_positions_y, lastPositionsY, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, accelerationX, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_y, accelerationY, numElements * sizeof(float), hipMemcpyHostToDevice);



    hipMemset(collided, 0, numElements * sizeof(bool));

    dim3 blockSize(16, 16);
    dim3 gridSize((numElements + blockSize.x - 1) / blockSize.x, (numElements + blockSize.y - 1) / blockSize.y);

    float blockSize2 = 256;
    int gridSize2 = (numElements + blockSize2 - 1) / blockSize2;
    float sub_dt = dt / (float)substeps;

    for (int j = 0; j < substeps; j++) {
        hipMemcpy(d_result_x, d_curr_positions_x, numElements * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_result_y, d_curr_positions_y, numElements * sizeof(float), hipMemcpyDeviceToDevice);
        solveContact<<<gridSize, blockSize>>>(d_curr_positions_x, d_curr_positions_y, d_result_x, d_result_y,
                                              d_radii, numElements, collided);
        hipDeviceSynchronize();

        hipMemcpy(d_curr_positions_x, d_result_x, numElements * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_curr_positions_y, d_result_y, numElements * sizeof(float), hipMemcpyDeviceToDevice);

        update<<<gridSize2, blockSize2>>>(d_curr_positions_x, d_curr_positions_y, d_last_positions_x,
                                          d_last_positions_y, d_acceleration_x, d_acceleration_y,
                                          xGravity, yGravity, sub_dt,
                                          numElements, d_radii, collided);
        hipDeviceSynchronize();
        hipMemset(collided, 0, numElements * sizeof(bool));
    }



    hipMemcpy(currPositionsX, d_curr_positions_x, numElements * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(currPositionsY, d_curr_positions_y, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(lastPositionsX, d_last_positions_x, numElements * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(lastPositionsY, d_last_positions_y, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(accelerationX, d_acceleration_x, numElements * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(accelerationY, d_acceleration_y, numElements * sizeof(float), hipMemcpyDeviceToHost);


    hipFree(d_curr_positions_x);
    hipFree(d_curr_positions_y);
    hipFree(d_radii);
    hipFree(d_result_x);
    hipFree(d_result_y);

    hipFree(d_last_positions_x);
    hipFree(d_last_positions_y);

    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
}
